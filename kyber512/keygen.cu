#include "hip/hip_runtime.h"
#include "keygen.h"



__global__ void keyGenKer(int16_t *t, int16_t *matrix_A, int32_t *sBar, int32_t *eBar,
						  int8_t *s,  int8_t *e, uint8_t *seed)
{
	//32 kB sahred memory 16*2*4*256, with SKEW_CHAR
	//__shared__ int8_t shmem[32*4][KYBER_N + SKEW_CHAR];

	int tid = blockDim.x*blockIdx.x + threadIdx.x;
	uint8_t rho_sigma[64];

	//hash_G
	SHA3_512Dev(rho_sigma, seed + KYBER_SEED_LEN * tid, KYBER_SEED_LEN);

	//Gen matrix A
	GenMatrixADev((int16_t *)(matrix_A + tid*(KYBER_K*KYBER_K*KYBER_N)), (uint8_t *)rho_sigma);

	//Gen vector s
	uint8_t nounce=0;
	for(int i=0; i<KYBER_K; ++i)
	{
		GensreDev(s + tid *KYBER_K*KYBER_N, rho_sigma+32, nounce);
		++nounce;
	}

	//Gen vector e
	for(int i=0; i<KYBER_K; ++i)
	{
		GensreDev(e + tid *KYBER_K*KYBER_N, rho_sigma+32, nounce);
		++nounce;
	}
	__syncthreads();

	//Raw NTT for the vectors
	nttVecI8Ker(sBar, s, GTab);

	nttVecI8Ker(eBar, e, GTab);

	//tBar = ABar*sBar + eBar
	for(int i=0; i<KYBER_K; ++i)
	{

	}
	for(int i=0; i<KYBER_N; ++i)
	{
		int64_t t_tmp = *()
	}

}

void keyGenCKer(uint8_t *pk, uint8_t *sk)
{

}

void testKeyGenCKer()
{
	uint8_t *h_seed = (uint8_t *)malloc(KYBER_SEED_LEN * TEST_CASES);


	uint8_t *d_seed;
	uint8_t *d_matrix_A;
	uint8_t *d_s;

	hipMalloc(&d_seed, KYBER_SEED_LEN * TEST_CASES);
	hipMalloc(&d_matrix_A, KYBER_N * KYBER_K * KYBER_K * TEST_CASES);
	hipMalloc(&d_s, KYBER_N * KYBER_K * TEST_CASES);

	//Generate seed d
	randombytes(h_seed, KYBER_SEED_LEN*TEST_CASES);
	hipMemcpy(d_seed, h_seed, KYBER_SEED_LEN*TEST_CASES, hipMemcpyHostToDevice);

	//Call Kernel
	keyGenKer(d_matrix_A, d_s, d_seed);

	free(h_seed);
	hipFree(d_seed);
	hipFree(d_matrix_A);
	hipFree(d_s);
}
