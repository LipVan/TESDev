#include "hip/hip_runtime.h"
#include "common.h"
using namespace nvcuda;


/*
 * CBD: Convert 128 bytes to 256 coefficients
 */

__device__ void CBDEtaDev(int8_t *out_poly, uint8_t *in_B)
{
	uint8_t tmp;
	int8_t a;
	int8_t b;

	for(int i=0; i<64*KYBER_ETA; i++)
	{
		for(int j=0; j<KYBER_ETA; j++)
		{
			tmp = (in_B[i] >> (1-j)*4) & 0xf;
			a = (tmp & 0x8)>>3 + (tmp & 0x4)>>2;
			b = (tmp & 0x2)>>1 + (tmp & 0x1);

			out_poly[i*KYBER_ETA+j] = (int8_t)(a-b);
		}
	}
}
/*
 * Parse: B* -> Rqn
 * Uniform sampling in Rq
 * */
__device__ int ParseDev(int16_t *out_poly, uint req_len, uint8_t *bytes, uint byte_len)
{
	uint16_t d;
	int i=0;
	int ctr=0;

	while(ctr < req_len && i+2<= byte_len)
	{
		d = ((uint16_t)bytes[i+1] << 8) | bytes[i];
		if(d < NINETEEN_Q)
		{
			d -= (d >> 12) * KYBER_Q;
			out_poly[ctr++] = (int16_t)d;
		}
		i += 2;
	}

	return ctr;
}


__device__ void GenMatrixADev(int16_t *matrixA, uint8_t *rho)
{
	uint64_t state[25]={0};
	uint8_t exseed[KYBER_SEED_LEN + 2];
	int ctr=0;
	uint8_t output[XOF_BLOCKBYTES];

	for(int i=0; i<KYBER_SEED_LEN; i++) exseed[i] = rho[i];

	for(int i=0; i<KYBER_K; i++)
	{
		for(int j=0; j<KYBER_K; j++)
		{
			int16_t *tmp_ptr = matrixA + (i*2+j)*KYBER_N;

			exseed[KYBER_SEED_LEN] = (uint8_t)j;
			exseed[KYBER_SEED_LEN+1] = (uint8_t)i;

			keccak1600AbsorbDev(state, SHAKE128_RATE, exseed, KYBER_SEED_LEN+2, 0x1F);

			keccak1600SqueezeDev(output, 1, state, SHAKE128_RATE);

			ctr = ParseDev(tmp_ptr, KYBER_N, output, XOF_BLOCKBYTES);

			while(ctr < KYBER_N)
			{
				keccak1600SqueezeDev(output, 1, state, SHAKE128_RATE);
				ctr += ParseDev(tmp_ptr+ctr, KYBER_N-ctr, output, XOF_BLOCKBYTES);
			}
		}
	}
}

__device__ void GensreDev(int8_t *vec, uint8_t *seed, uint8_t nounce)
{
	uint8_t exseed[KYBER_SEED_LEN+1];
	uint8_t output[KYBER_ETA*KYBER_N/4];

	for(int i=0; i<KYBER_SEED_LEN; i++) exseed[i] = seed[i];

	for(int i=0; i<KYBER_K; ++i)
	{
		exseed[KYBER_SEED_LEN] = i;
		uint64_t state[25]={0};

		keccak1600AbsorbDev(state, SHAKE256_RATE, exseed, KYBER_SEED_LEN+1, 0x1F);
		keccak1600SqueezeDev(output, KYBER_ETA*KYBER_N/4/XOF_BLOCKBYTES, state, SHAKE256_RATE);

		CBDEtaDev(vec, output);
	}
}


//Signed char ->16x16x16
//__global__ void nttVecI8Ker(int16_t *o_vec, int8_t *i_vec, int8_t *i_tabg)
//{
//	//Perhaps the shared memory is not enough for KYBER_N*KYBER_N = 64 kB
////	__shared__ int8_t shmem[KYBER_N*KYBER_N];
//
//	//Copy data from global memory to shared memory
//
//	int warp_id = threadIdx.x / WARP_SIZE;
//
//	wmma::fragment<wmma::matrix_a, WMM_M, WMM_N, WMM_K, char, wmma::row_major> a_frag;
//	wmma::fragment<wmma::matrix_b, WMM_M, WMM_N, WMM_K, char, wmma::row_major> b_frag;
//	wmma::fragment<wmma::accumulator, WMM_M, WMM_N, WMM_K, int> c_frag[4];
//
//	for(int i=0; i<4; ++i)
//	{
//		wmma::fill_fragment(c_frag[i], 0);
//	}
//
//	//c0 = a0 * b0
//	for(int round=0; round<KYBER_N/WMM_N; ++round)
//	{
//		int8_t *a_warp_ptr = (int8_t *)i_vec + round*WMM_N;
//
//		wmma::load_matrix_sync(a_frag, a_warp_ptr, KYBER_N);
//
//#pragma unroll
//		for(int ind=0; ind<4; ++ind)
//		{
//			int8_t *b_warp_ptr = (warp_id < WARPS_PER_BLOCK/2)? (i_tabg + round*WMM_N*KYBER_N + (warp_id*2 + ind)*WMM_K):
//																(i_tabg + TABLE_OFFSET + round*WMM_N*KYBER_N + (warp_id*2 + ind)*WMM_K);
//
//			wmma::load_matrix_sync(b_frag, b_warp_ptr , KYBER_N);
//			wmma::mma_sync(c_frag[ind], a_frag, b_frag, c_frag[ind]);
//		}
//	}
//}

__global__ void nttVecI8Ker(int32_t *o_vec, int8_t *i_vec, int8_t *i_tabg)
{
	//Perhaps the shared memory is not enough for KYBER_N*KYBER_N = 64 kB
//	__shared__ int8_t shmem[KYBER_N*KYBER_N];

	//Copy data from global memory to shared memory

	int warp_id = threadIdx.x / WARP_SIZE;

	wmma::fragment<wmma::matrix_a, WMM_M, WMM_N, WMM_K, char, wmma::row_major> a_frag;
	wmma::fragment<wmma::matrix_b, WMM_M, WMM_N, WMM_K, char, wmma::row_major> b_frag;
	wmma::fragment<wmma::accumulator, WMM_M, WMM_N, WMM_K, int> c_frag[2];

	for(int i=0; i<2; ++i)
	{
		wmma::fill_fragment(c_frag[i], 0);
	}

	//c0 = a0 * b0
	for(int round=0; round<KYBER_N/WMM_N; ++round)
	{
		int8_t *a_warp_ptr = (int8_t *)i_vec + round*WMM_N;

		wmma::load_matrix_sync(a_frag, a_warp_ptr, KYBER_N);

#pragma unroll
		for(int ind=0; ind<2; ++ind)
		{
			int8_t *b_warp_ptr = i_tabg + round*WMM_N*KYBER_N + (warp_id*2 + ind)*WMM_K;

			wmma::load_matrix_sync(b_frag, b_warp_ptr , KYBER_N);
			wmma::mma_sync(c_frag[ind], a_frag, b_frag, c_frag[ind]);
		}
	}

	//c0 <-- c0*BASE_BITS
	for(int ind=0; ind<2; ++ind)
	{
		for(int t=0; t<c_frag[ind].num_elements; ++t)
		{
			c_frag[ind].x[t] *= BASE_BITS;
		}
	}

	//c1 <- a1 * b1 + c0
	for(int round=0; round<KYBER_N/WMM_N; ++round)
	{
		int8_t *a_warp_ptr = (int8_t *)i_vec + round*WMM_N;
		wmma::load_matrix_sync(a_frag, a_warp_ptr, KYBER_N);

#pragma unroll
		for(int ind=0; ind<2; ++ind)
		{
			int8_t *b_warp_ptr = (int8_t *)i_tabg + TABLE_OFFSET + round*WMM_N*KYBER_N + (warp_id*2 + ind)*WMM_K;

			wmma::load_matrix_sync(b_frag, b_warp_ptr, KYBER_N);
			wmma::mma_sync(c_frag[ind], a_frag, b_frag, c_frag[ind]);
		}
	}

	//Store the result
	for(int ind=0; ind<2; ++ind)
	{
		int *o_warp_ptr = o_vec + (warp_id*2 + ind)*WMM_K;

		wmma::store_matrix_sync(o_warp_ptr, c_frag[ind], KYBER_N);
	}
//	__syncthreads();
}
